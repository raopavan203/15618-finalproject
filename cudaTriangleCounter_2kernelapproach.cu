#include "hip/hip_runtime.h"
/*
 * Triangle counter with workload balancing
 *
 * @author: Manish Jain
 * @author: Vashishtha Adtani
 */

#include <iostream>
#include <string>
#include <sstream>
#include <algorithm>
#include <stdlib.h>
#include <math.h>
#include <stdio.h>
#include <vector>
#include <thrust/scan.h>                                                        
#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>

#include <fstream>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include "cudaTriangleCounter.h"

#define BLOCK_SIZE 32

struct GlobalConstants {

    int *NodeList;
    int *ListLen;
    int numNodes;
    int numEdges;
};

__constant__ GlobalConstants cuConstCounterParams;

void
CudaTriangleCounter::setup() {

    int deviceCount = 0;
    std::string name;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Initializing CUDA for CountingTriangles\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        name = deviceProps.name;

        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n", static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);

    }
    printf("---------------------------------------------------------\n");

    // By this time the graph should be loaded.  Copying graph to 
    // data structures into device memory so that it is accessible to
    // CUDA kernels
    //

    hipMalloc(&cudaDeviceListLen, sizeof(int ) * numNodes);
    hipMemcpy(cudaDeviceListLen, list_len, sizeof(int) * numNodes, hipMemcpyHostToDevice);

    hipMalloc((void **)&cudaDeviceNodeList, node_list_size * sizeof(int));
    hipMemcpy(cudaDeviceNodeList, node_list, sizeof(int) * node_list_size, hipMemcpyHostToDevice);

    GlobalConstants params;
    params.ListLen = cudaDeviceListLen;
    params.NodeList = cudaDeviceNodeList;
    params.numNodes = numNodes;
    params.numEdges = numEdges;
    hipMemcpyToSymbol(HIP_SYMBOL(cuConstCounterParams), &params, sizeof(GlobalConstants));
}

CudaTriangleCounter::CudaTriangleCounter(char *fileName) {
    clock_t start, diff, malloc_diff;
    int node, edge_id, temp = 0;
    int total_nodes = 0;
    int total_edges = 0;
    int msec;

    std::string line;
    std::ifstream myfile;
    myfile.open(fileName);

    std::string token;                                                             
    if (strstr(fileName,"new_orkut") != NULL) {                                    
        printf("This is the NEW_ORKUT FILE **\n");                             
        total_nodes = 3072600;                                                     
        total_edges = 117185083 + 1;                                               
    } else {                                                                       
        std::getline(myfile,line);                                                 
        std::stringstream lineStream(line);                                        
        while (lineStream >> token) {                                              
            if (temp == 0) {                                                       
                total_nodes = std::stoi(token, NULL, 10) + 1;                      
            } else if (temp == 1) {                                                
                total_edges = std::stoi(token, NULL, 10) + 1;                      
            } else {                                                               
                printf("!!!!!!!!!!!! TEMP IS %d\n ", temp);                        
                break;                                                             
            }                                                                      
            temp++;                                                                
        }                                                                          
    }

    start = clock();

    numNodes = total_nodes;
    node_list_size = total_edges * 2;
    numEdges = total_edges;

    printf("total_nodes %d\n", total_nodes);
    printf("node_list_size %d\n", node_list_size);
    printf("numEdges %d\n", numEdges);

    list_len = (int *)calloc(total_nodes, sizeof(int));
    start_addr = (int *)calloc(total_nodes, sizeof(int));
    node_list = (int *)calloc(node_list_size, sizeof(int));

    malloc_diff = clock() - start;
    msec = malloc_diff * 1000 / CLOCKS_PER_SEC;

    printf("memory allocated ......\n");
    node = 1;
    temp = 1;
    int neighbors;
    while(std::getline(myfile, line)) {
        neighbors = 0;
        std::stringstream lineStream(line);
        std::string token;
        while(lineStream >> token)
        {
            edge_id = std::stoi(token, NULL, 10);
            if (edge_id > node) {
                node_list[temp++] = edge_id;
                neighbors++;
            }
        }

        list_len[node] = neighbors;
        node++;
    }

    printf("graph created......\n");
    diff = clock() - start;
    msec = diff * 1000 / CLOCKS_PER_SEC;
    printf("time taken %d seconds %d milliseconds\n", msec/1000, msec%1000);

    myfile.close();
}

CudaTriangleCounter::~CudaTriangleCounter() {

    free(node_list);
    free(list_len);
}


/*
 * Kernel to count number of triangles formed by a single edge. And store the count
 * in an array on which we will run reduction later to find total number of triangles
 * in the given graph.
 */
__global__ void countTriangleKernel(int *countArray, edge_tuple_t *compressed_list, int *start_addr, int num) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= num) {
        return;
    }

    if (i == 0) {
        countArray[i] = 0;
        return;
    }

    int j = 0, k = 0, count=0;
    int *node_list = cuConstCounterParams.NodeList;
    int *list_len = cuConstCounterParams.ListLen;
    edge_tuple_t *edgeList = compressed_list;

    int u = edgeList[i].u;
    int v = edgeList[i].v;

    /* Fetching neigbour vertices from the node list */
    int *list1 = node_list + start_addr[u-1] + 1;
    int len1 = list_len[u];

    int *list2 = node_list + start_addr[v-1] + 1;
    int len2 = list_len[v];

    /* 
     * Traversing both lists to find the common nodes. Each common node
     * will be counted as a triangle
     */
    while ( j < len1 && k < len2) {

        if (list1[j] == list2[k]) {
            count++;
            j++;
            k++;
        } else if (list1[j] < list2[k]) {
            j++;
        } else {
            k++;
        }
    }

    countArray[i] = count;
}


/*
 * Creating data structure which stores all the edges
 */
__global__ void createEdgeList(edge_tuple_t *edge_list, int *start_addr) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= cuConstCounterParams.numNodes) {
        return;
    }

    if (i == 0) {
        return;
    }

    int *node_list = cuConstCounterParams.NodeList;
    int *list_len = cuConstCounterParams.ListLen;
    int start_index = start_addr[i-1] + 1;
    int *list = node_list + start_addr[i-1] + 1;
    int len = list_len[i];

    for (int j=0; j<len; j++) {
        edge_list[start_index].u = i;
        edge_list[start_index].v = list[j];
        start_index++;
    }
}

#define THRESHOLD 50000

__global__ void segregateList(edge_tuple_t *edge_list, int *small_edge, int *large_edge) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int *list_len = cuConstCounterParams.ListLen;

    if ( i >= cuConstCounterParams.numEdges) {
        return;
    }

    if (i == 0) {
        large_edge[i] = 0;
        small_edge[i] = 0;
        return;
    }

    int u = edge_list[i].u;
    int v = edge_list[i].v;

    if ((list_len[u] > THRESHOLD) || (list_len[v] > THRESHOLD)) {
        large_edge[i] = 1;
        small_edge[i] = 0;
    } else {
        large_edge[i] = 0;
        small_edge[i] = 1;
    }
}

__global__ void createSmallList(edge_tuple_t *edge_list, edge_tuple_t *small_edge_list, int *small_edge) { 
 
    int i = blockIdx.x * blockDim.x + threadIdx.x; 

    if ( i >= (cuConstCounterParams.numEdges)) {
        return;
    }

    if (small_edge[i] != small_edge[i+1]) {
        int index = small_edge[i];
        small_edge_list[index] = edge_list[i];
    }
}

__global__ void createLargeList(edge_tuple_t *edge_list, edge_tuple_t *large_edge_list, int *large_edge) { 
 
    int i = blockIdx.x * blockDim.x + threadIdx.x; 

    if ( i >= (cuConstCounterParams.numEdges-1)) {
        return;
    }

    if (large_edge[i] != large_edge[i+1]) {
        int index = large_edge[i];
        large_edge_list[index] = edge_list[i];
    }
}


/*
 * Counts the number of triangles in the given graph. We first find out the
 * starting address of each list where list stores the neighbours of particular
 * node. We then create the list of all edges from the given nodes and their
 * neighbours.
 */
void
CudaTriangleCounter::countTriangles() {

    dim3 blockdim  = BLOCK_SIZE;
    dim3 griddim = (numEdges + BLOCK_SIZE)/BLOCK_SIZE;
    dim3 griddim1 = (numNodes + BLOCK_SIZE)/BLOCK_SIZE;
    int count;
    edge_tuple_t *edge_list, *small_edge_list, *large_edge_list;
    int *small_edge, *large_edge;
    int num_small_edges, num_large_edges;
    int *temp;

    /* Calculating start address of each neighbour list */
    hipMalloc(&cudaDeviceStartAddr, sizeof(int ) * numNodes);
    thrust::device_ptr<int> dev_ptr1(cudaDeviceListLen);
    thrust::device_ptr<int> output_ptr(cudaDeviceStartAddr);
    thrust::inclusive_scan(dev_ptr1, dev_ptr1 + numNodes, output_ptr);

    /* Create a list of all edges present in the graph */
    hipMalloc((void **)&edge_list, numEdges * sizeof(edge_tuple_t));
    createEdgeList<<<griddim1, blockdim>>>(edge_list, cudaDeviceStartAddr);
    hipDeviceSynchronize();

    hipMalloc(&small_edge, sizeof(int ) * numEdges);
    hipMalloc(&large_edge, sizeof(int ) * numEdges);

    segregateList<<<griddim, blockdim>>>(edge_list, small_edge, large_edge);
    hipDeviceSynchronize();

    thrust::device_ptr<int> small_ptr(small_edge);
    thrust::inclusive_scan(small_ptr, small_ptr + numEdges, small_ptr);

    thrust::device_ptr<int> large_ptr(large_edge);
    thrust::inclusive_scan(large_ptr, large_ptr + numEdges, large_ptr);

    temp = (int *) malloc (numEdges * sizeof(int));
    hipMemcpy(temp, small_edge, sizeof(int) * numEdges, hipMemcpyDeviceToHost);

    hipMemcpy(&num_small_edges, &small_edge[numEdges-1], sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&num_large_edges, &large_edge[numEdges-1], sizeof(int), hipMemcpyDeviceToHost);
    hipMalloc((void **)&small_edge_list, ( 1 +num_small_edges) * sizeof(edge_tuple_t));
    hipMalloc((void **)&large_edge_list, ( 1 + num_large_edges) * sizeof(edge_tuple_t));

    createSmallList<<<griddim, blockdim>>>(edge_list, small_edge_list, small_edge);
    hipDeviceSynchronize();

    createLargeList<<<griddim, blockdim>>>(edge_list, large_edge_list, large_edge);
    hipDeviceSynchronize();

    int *countArraySmall, *countArrayLarge;
    
    hipMalloc((void **)&countArraySmall, (2 + num_small_edges) * sizeof(int));
    hipMalloc((void **)&countArrayLarge, (2 + num_large_edges) * sizeof(int));

    dim3 griddim2 = (num_small_edges + 1 + BLOCK_SIZE)/BLOCK_SIZE;

    /* Applying intersection rule on all small edges to find number of triangles */
    countTriangleKernel<<<griddim2, blockdim>>>(countArraySmall, small_edge_list, cudaDeviceStartAddr, num_small_edges+1);
    hipDeviceSynchronize();

    thrust::device_ptr<int> dev_ptr2(countArraySmall);
    thrust::inclusive_scan(dev_ptr2, dev_ptr2 + num_small_edges+1, dev_ptr2);

    int count1, count2;
    hipMemcpy(&count1, &countArraySmall[num_small_edges], sizeof(int), hipMemcpyDeviceToHost);


    dim3 griddim3 = (num_large_edges + 1 + BLOCK_SIZE)/BLOCK_SIZE;

    /* Applying intersection rule on all large edges to find number of triangles */
    countTriangleKernel<<<griddim3, blockdim>>>(countArrayLarge, large_edge_list, cudaDeviceStartAddr, num_large_edges+1);
    hipDeviceSynchronize();

    thrust::device_ptr<int> dev_ptr3(countArrayLarge);
    thrust::inclusive_scan(dev_ptr3, dev_ptr3 + num_large_edges + 1, dev_ptr3);
    hipMemcpy(&count2, &countArrayLarge[num_large_edges], sizeof(int), hipMemcpyDeviceToHost);

    count = count1 + count2;
    printf("count %d\n", count);
}

